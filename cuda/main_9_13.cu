#include "hip/hip_runtime.h"
#include <iostream>
#include <bit>
#include <bitset>
#include "../Data/Data_9_13_6.cpp"

#define NBR_RIDGES 1210 //first multiple of 220 larger than 1190
#define NBR_LOOPS 121 //out of 121
#define RESULT_SIZE (1u<<25)
#define SUB_BLOCK 4
#define DIVISOR (32/SUB_BLOCK)
#define BLOCK_SIZE 110

using namespace std;

struct StructX0 {
    unsigned long X0 = 0ul;
    unsigned int precalc[14] = {0u};
};
const int nbrX0 = NBR_X0;
const int nbrX1 = NBR_X1;
__shared__ int r[NBR_RIDGES];
__device__ __managed__  unsigned int ai[SUB_BLOCK][BLOCK_SIZE];
__device__ __managed__  int mi[SUB_BLOCK][N][BLOCK_SIZE];
__device__ __managed__  unsigned int listX1[nbrX1];
__device__ __managed__  unsigned long out[RESULT_SIZE];
__device__ __managed__  int nOut = 0;
__device__ __managed__  StructX0 listX0[nbrX0];

__global__ void kernel(StructX0 structX0[]) {
    unsigned int a[SUB_BLOCK];
    unsigned int precalc_a = structX0[blockIdx.x].precalc[threadIdx.x / DIVISOR];
    unsigned long X0 = structX0[blockIdx.x].X0;
    for (int k = 0; k < SUB_BLOCK; k++) {
        a[k] = ai[k][threadIdx.x] | (((precalc_a >> (SUB_BLOCK * (threadIdx.x % DIVISOR) + k)) & 1u) << 31);
    }
    int m[SUB_BLOCK][N];
    for (int k = 0; k < SUB_BLOCK; k++) {
        for (int l = 0; l < N; l++) {
            m[k][l] = mi[k][l][threadIdx.x];
        }
    }
    int count;
    bool Ax[SUB_BLOCK];
    bool stop;
    for (unsigned int X1: listX1) {
        stop = false;
        for (int i = 0; i < NBR_RIDGES; i += BLOCK_SIZE) r[i + threadIdx.x] = 0;
        __syncthreads();
        for (int j = 0; j < SUB_BLOCK; j++) {
            Ax[j] = __popc(a[j] & X1) & 1;
        }
        count = 0;
        for (bool j: Ax) {
            count += __syncthreads_count(j);
        }
        if (count > MAX_NBR_FACETS) continue;
        for (int k = 0; k < SUB_BLOCK; k++) {
            if (stop) break;
            if (Ax[k]) {
                for (int t = 0; t < N; t++) {
                    if (atomicAdd(r + m[k][t], 1) >= 2) {
                        stop = true;
                        break;
                    }
                }
            }
        }
        if (__syncthreads_or(stop)) continue;
        if (threadIdx.x == 0) {
            out[atomicAdd(&nOut, 1)] = (X0 | (unsigned long) (X1 ^ (1u << 31)));
        }
    }

}

void increment_vect(unsigned int vect[], const unsigned int ref[], const int starting_index, const int size) {
    vect[0] = (vect[0] + 1) % ref[starting_index];
    int k = 0;
    while (vect[k] == 0 and k < size - 1) {
        k += 1;
        vect[k] = (vect[k] + 1) % ref[starting_index + k];
    }
}


int main() {
    unsigned int vectX0[sizeVectX0];
    for (int k = 0; k < sizeVectX0; k++) vectX0[k] = 0;
    unsigned int vectX1[sizeVectX1];
    for (int k = 0; k < sizeVectX1; k++) vectX1[k] = 0;
    unsigned int list_shifts[NBR_GROUPS];
    unsigned int list_ref[NBR_GROUPS - 1];
    unsigned long list_elementary[NBR_GROUPS - 1][11];
    unsigned long X0;
    //Initialiser les matrices ai et mi
    for (int k = 0; k < NBR_FACETS; k++) {
        ai[k % SUB_BLOCK][k / SUB_BLOCK] = ((A[k] << 33) >> 33);
    }
    for (int k = 0; k < NBR_FACETS; k++) {
        for (int l = 0; l < N; l++) {
            mi[k % SUB_BLOCK][l][k / SUB_BLOCK] = M[l][k];
        }
    }
    //Initialiser les shifts et les générateurs de combinaison linéaire
    list_shifts[NBR_GROUPS - 1] = 64 - NBR_GENERATORS;
    for (int k = NBR_GROUPS - 2; k > -1; k--) {
        list_shifts[k] = list_groups[k + 1] + list_shifts[k + 1];
    }
    for (int i = 1; i < NBR_GROUPS; i++) {
        int position = 0;
        for (int j = 0; j < (1ul << (list_groups[i])); j++) {
            if (__popcount(j) <= 2) {
                unsigned long jl = j;
                list_elementary[i - 1][position] = (jl << list_shifts[i]);
                position += 1;
            }
        }
        list_ref[i - 1] = position;
    }
    //Initialiser les listX1
    unsigned int X1;
    for (unsigned int &X1_val: listX1) {
        X1 = 1u << 31;
        for (int i = 0; i < sizeVectX1; i++) {
            X1 |= (list_elementary[i + sizeVectX0][vectX1[i]]);
        }
        X1_val = X1;
        increment_vect(vectX1, list_ref, sizeVectX0, sizeVectX1);
    }
    bool last_one_copied = false;
    bool first_appeared;
    for (int l = 0; l < NBR_LOOPS; l++) {
        for (auto & dataX0 : listX0) {
            for (unsigned int &dataPrecalc:dataX0.precalc){
                dataPrecalc=0;
            }
        }
        last_one_copied = false;
        for (auto & dataX0 : listX0) {
            X0 = (1ul << 63);
            for (int i = 0; i < sizeVectX0; i++) {
                X0 |= list_elementary[i][vectX0[i]];
            }
            dataX0.X0 = X0;
            increment_vect(vectX0, list_ref, 0, sizeVectX0);
        }
        for (auto & dataX0 : listX0) {
            for (int i = 0; i < BLOCK_SIZE; i++) {
                for (int k = 0; k < SUB_BLOCK; k++) {
                    if ((__popcount(dataX0.X0 & A[i * SUB_BLOCK + k])) & 1u) {
                        dataX0.precalc[i / DIVISOR] |= (1u << (SUB_BLOCK * (i % DIVISOR) + k));
                    }
                }
            }
        }
        kernel<<<NBR_X0, BLOCK_SIZE>>>(listX0);
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess)
            printf("kernel launch failed with error \"%s\".\n",
                   hipGetErrorString(cudaerr));
        if (nOut > (1u << 23)) {
            for (int i = 0; i < nOut; i++) {
                first_appeared = false;
                cout << '[';
                for (int j = 0; j < NBR_FACETS; j++) {
                    if (__popcount(out[i] & A[j]) & 1ul) {
                        if (first_appeared) cout << ',';
                        first_appeared = true;
                        cout << F[j];
                    }
                }
                cout << ']' << '\n';
            }
            nOut = 0;
            last_one_copied = true;
        }

    }
    if (not last_one_copied) {
        for (int i = 0; i < nOut; i++) {
            cout << '[';
            first_appeared = false;
            for (int j = 0; j < NBR_FACETS; j++) {
                if (__popcount(out[i] & A[j]) & 1ul) {
                    if (first_appeared) cout << ',';
                    first_appeared = true;
                    cout << F[j];
                }
            }
            cout << ']' << '\n';
        }
    }
    return 0;
}