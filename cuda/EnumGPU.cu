
#include <hip/hip_runtime.h>
#include <iostream>
#include <bit>
#include <bitset>
#include <cstdio>
// Compile this script to test if you have correctly installed CUDA Toolkit
int main(){
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    int device;
    for (device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("Device %d has compute capability %d.%d.\n",
               device, deviceProp.major, deviceProp.minor);
    }
}
